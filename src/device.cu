#include "hip/hip_runtime.h"
// device.cu -- GPU specific operations utilizing the CUDA API.
#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include "vector_arithmetic.h"	// for arbitrary prec. vectors
//#include <vector_functions.h>	// for single prec. vectors
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

#include "sphere.h"
#include "datatypes.h"
#include "utility.h"
#include "constants.cuh"
#include "debug.h"

#include "sorting.cuh"	
#include "contactmodels.cuh"
#include "cohesion.cuh"
#include "contactsearch.cuh"
#include "integration.cuh"
#include "raytracer.cuh"
#include "navierstokes.cuh"

// Returns the number of cores per streaming multiprocessor, which is
// a function of the device compute capability
int cudaCoresPerSM(int major, int minor)
{
    if (major == 1)
        return 8;
    else if (major == 2 && minor == 0)
        return 32;
    else if (major == 2 && minor == 1)
        return 48;
    else if (major == 3 && minor == 0)
        return 192;
    else if (major == 3 && minor == 5)
        return 192;
    else if (major == 5 && minor == 0)
        return 128;
    else
        printf("Error in cudaCoresPerSM",
               "Device compute capability value (%d.%d) not recognized.",
               major, minor);
    return -1;
}

// Wrapper function for initializing the CUDA components.
// Called from main.cpp
__host__ void DEM::initializeGPU(void)
{
    using std::cout; // stdout

    // Specify target device
    int cudadevice = 0;

    // Variables containing device properties
    hipDeviceProp_t prop;
    int deviceCount;
    int cudaDriverVersion;
    int cudaRuntimeVersion;

    checkForCudaErrors("Before initializing CUDA device");

    // Register number of devices
    hipGetDeviceCount(&deviceCount);
    ndevices = deviceCount; // store in DEM class

    if (deviceCount == 0) {
        std::cerr << "\nERROR: No CUDA-enabled devices availible. Bye."
            << std::endl;
        exit(EXIT_FAILURE);
    } else if (deviceCount == 1) {
        if (verbose == 1)
            cout << "  System contains 1 CUDA compatible device.\n";
    } else {
        if (verbose == 1)
            cout << "  System contains " << deviceCount
                << " CUDA compatible devices.\n";
    }

    // Loop through GPU's and choose the one with the most CUDA cores
    int ncudacores;
    int max_ncudacores = 0;
    for (int d=0; d<ndevices; d++) {
        hipGetDeviceProperties(&prop, d);
        hipDriverGetVersion(&cudaDriverVersion);
        hipRuntimeGetVersion(&cudaRuntimeVersion);

        ncudacores = prop.multiProcessorCount
            *cudaCoresPerSM(prop.major, prop.minor);
        if (ncudacores > max_ncudacores) {
            max_ncudacores = ncudacores;
            cudadevice = d;
        }

        if (verbose == 1) {
            cout << "  CUDA device ID: " << d << "\n";
            cout << "  - Name: " <<  prop.name << ", compute capability: " 
                << prop.major << "." << prop.minor << ".\n";
            cout << "  - CUDA Driver version: " << cudaDriverVersion/1000 
                << "." <<  cudaDriverVersion%100 
                << ", runtime version " << cudaRuntimeVersion/1000 << "." 
                << cudaRuntimeVersion%100 << std::endl;
        }
    }

    device = cudadevice; // store in DEM class
    cout << " Using CUDA device ID " << cudadevice << " with "
         << max_ncudacores << " cores." << std::endl;

    // Comment following line when using a system only containing
    // exclusive mode GPUs
    hipChooseDevice(&cudadevice, &prop);

    checkForCudaErrors("While initializing CUDA device");
}

// Start timer for kernel profiling
__host__ void startTimer(hipEvent_t* kernel_tic)
{
    hipEventRecord(*kernel_tic);
}

// Stop timer for kernel profiling and time to function sum
__host__ void stopTimer(hipEvent_t *kernel_tic,
        hipEvent_t *kernel_toc,
        float *kernel_elapsed,
        double* sum)
{
    hipEventRecord(*kernel_toc, 0);
    hipEventSynchronize(*kernel_toc);
    hipEventElapsedTime(kernel_elapsed, *kernel_tic, *kernel_toc);
    *sum += *kernel_elapsed;
}

// Check values of parameters in constant memory
__global__ void checkConstantValues(int* dev_equal,
        Grid* dev_grid,
        Params* dev_params)
{
    // Values ok (0)
    *dev_equal = 0;

    // Compare values between global- and constant
    // memory structures
    if (dev_grid->origo[0] != devC_grid.origo[0] ||
            dev_grid->origo[1] != devC_grid.origo[1] ||
            dev_grid->origo[2] != devC_grid.origo[2] ||
            dev_grid->L[0] != devC_grid.L[0] ||
            dev_grid->L[1] != devC_grid.L[1] ||
            dev_grid->L[2] != devC_grid.L[2] ||
            dev_grid->num[0] != devC_grid.num[0] ||
            dev_grid->num[1] != devC_grid.num[1] ||
            dev_grid->num[2] != devC_grid.num[2] ||
            dev_grid->periodic != devC_grid.periodic)
        *dev_equal = 1; // Not ok

    else if (dev_params->g[0] != devC_params.g[0] ||
            dev_params->g[1] != devC_params.g[1] ||
            dev_params->g[2] != devC_params.g[2] ||
            dev_params->k_n != devC_params.k_n ||
            dev_params->k_t != devC_params.k_t ||
            dev_params->k_r != devC_params.k_r ||
            dev_params->gamma_n != devC_params.gamma_n ||
            dev_params->gamma_t != devC_params.gamma_t ||
            dev_params->gamma_r != devC_params.gamma_r ||
            dev_params->mu_s != devC_params.mu_s ||
            dev_params->mu_d != devC_params.mu_d ||
            dev_params->mu_r != devC_params.mu_r ||
            dev_params->rho != devC_params.rho ||
            dev_params->contactmodel != devC_params.contactmodel ||
            dev_params->kappa != devC_params.kappa ||
            dev_params->db != devC_params.db ||
            dev_params->V_b != devC_params.V_b ||
            dev_params->lambda_bar != devC_params.lambda_bar ||
            dev_params->nb0 != devC_params.nb0 ||
            dev_params->mu != devC_params.mu ||
            dev_params->rho_f != devC_params.rho_f)
        *dev_equal = 2; // Not ok
}

// Copy the constant data components to device memory,
// and check whether the values correspond to the 
// values in constant memory.
__host__ void DEM::checkConstantMemory()
{
    // Allocate space in global device memory
    Grid* dev_grid;
    Params* dev_params;
    hipMalloc((void**)&dev_grid, sizeof(Grid));
    hipMalloc((void**)&dev_params, sizeof(Params));

    // Copy structure data from host to global device memory
    hipMemcpy(dev_grid, &grid, sizeof(Grid), hipMemcpyHostToDevice);
    hipMemcpy(dev_params, &params, sizeof(Params), hipMemcpyHostToDevice);

    // Compare values between global and constant memory
    // structures on the device.
    int* equal = new int;	// The values are equal = 0, if not = 1
    *equal = 0;
    int* dev_equal;
    hipMalloc((void**)&dev_equal, sizeof(int));
    checkConstantValues<<<1,1>>>(dev_equal, dev_grid, dev_params);
    checkForCudaErrors("After constant memory check");

    // Copy result to host
    hipMemcpy(equal, dev_equal, sizeof(int), hipMemcpyDeviceToHost);

    // Free global device memory
    hipFree(dev_grid);
    hipFree(dev_params);
    hipFree(dev_equal);

    // Are the values equal?
    if (*equal != 0) {
        std::cerr << "Error! The values in constant memory do not "
            << "seem to be correct (" << *equal << ")." << std::endl;
        exit(1);
    } else {
        if (verbose == 1)
            std::cout << "  Constant values ok (" << *equal << ")."
                << std::endl;
    }
}

// Copy selected constant components to constant device memory.
__host__ void DEM::transferToConstantDeviceMemory()
{
    using std::cout;

    if (verbose == 1)
        cout << "  Transfering data to constant device memory:     ";

    // Copy to main device
    hipMemcpyToSymbol(HIP_SYMBOL(devC_nd), &nd, sizeof(nd));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_np), &np, sizeof(np));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_nw), &walls.nw, sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_nc), &NC, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_dt), &time.dt, sizeof(Float));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_grid), &grid, sizeof(Grid));
    hipMemcpyToSymbol(HIP_SYMBOL(devC_params), &params, sizeof(Params));
    checkForCudaErrors("After transferring to device constant memory");

    // Allocate constant memory on helper devices
    hdevC_nd     = (unsigned*)malloc(ndevices*sizeof(unsigned));
    hdevC_np     = (unsigned*)malloc(ndevices*sizeof(unsigned));
    hdevC_nw     = (unsigned*)malloc(ndevices*sizeof(unsigned));
    hdevC_nc     = (int*)malloc(ndevices*sizeof(int));
    hdevC_dt     = (Float*)malloc(ndevices*sizeof(Float));
    hdevC_nb0    = (unsigned*)malloc(ndevices*sizeof(unsigned));
    hdevC_params = (Params*)malloc(ndevices*sizeof(Params));
    hdevC_grid   = (Grid*)malloc(ndevices*sizeof(Grid));

    // Copy to helper devices (and main device)
    for (int d=0; d<ndevices; d++) {
        hipSetDevice(d);

        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_nd[d]), &nd, sizeof(nd));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_np[d]), &np, sizeof(np));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_nw[d]), &walls.nw, sizeof(unsigned int));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_nc[d]), &NC, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_dt[d]), &time.dt, sizeof(Float));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_grid[d]), &grid, sizeof(Grid));
        hipMemcpyToSymbol(HIP_SYMBOL(hdevC_params[d]), &params, sizeof(Params));
        checkForCudaErrors("During transfer to helper device constant memory");
    }
    hipSetDevice(device);

    if (verbose == 1)
        cout << "Done\n";

    // only for main device
    checkConstantMemory();
}


// Allocate device memory for particle variables,
// tied to previously declared pointers in structures
__host__ void DEM::allocateGlobalDeviceMemory(void)
{
    // Particle memory size
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    if (verbose == 1)
        std::cout << "  Allocating global device memory:                ";

    k.acc = new Float4[np];
    k.angacc = new Float4[np];
#pragma omp parallel for if(np>100)
    for (unsigned int i = 0; i<np; ++i) {
        k.acc[i] = MAKE_FLOAT4(0.0, 0.0, 0.0, 0.0);
        k.angacc[i] = MAKE_FLOAT4(0.0, 0.0, 0.0, 0.0);
    }

    // Kinematics arrays
    hipMalloc((void**)&dev_x, memSizeF4);
    hipMalloc((void**)&dev_xyzsum, memSizeF4);
    hipMalloc((void**)&dev_vel, memSizeF4);
    hipMalloc((void**)&dev_vel0, memSizeF4);
    hipMalloc((void**)&dev_acc, memSizeF4);
    hipMalloc((void**)&dev_force, memSizeF4);
    hipMalloc((void**)&dev_angpos, memSizeF4);
    hipMalloc((void**)&dev_angvel, memSizeF4);
    hipMalloc((void**)&dev_angvel0, memSizeF4);
    hipMalloc((void**)&dev_angacc, memSizeF4);
    hipMalloc((void**)&dev_torque, memSizeF4);

    // Particle contact bookkeeping arrays
    hipMalloc((void**)&dev_contacts,
               sizeof(unsigned int)*np*NC);
    hipMalloc((void**)&dev_distmod, memSizeF4*NC);
    hipMalloc((void**)&dev_delta_t, memSizeF4*NC);
    hipMalloc((void**)&dev_bonds, sizeof(uint2)*params.nb0);
    hipMalloc((void**)&dev_bonds_delta, sizeof(Float4)*params.nb0);
    hipMalloc((void**)&dev_bonds_omega, sizeof(Float4)*params.nb0);

    // Sorted arrays
    hipMalloc((void**)&dev_x_sorted, memSizeF4);
    hipMalloc((void**)&dev_vel_sorted, memSizeF4);
    hipMalloc((void**)&dev_angvel_sorted, memSizeF4);

    // Energy arrays
    hipMalloc((void**)&dev_es_dot, memSizeF);
    hipMalloc((void**)&dev_ev_dot, memSizeF);
    hipMalloc((void**)&dev_es, memSizeF);
    hipMalloc((void**)&dev_ev, memSizeF);
    hipMalloc((void**)&dev_p, memSizeF);

    // Cell-related arrays
    hipMalloc((void**)&dev_gridParticleCellID, sizeof(unsigned int)*np);
    hipMalloc((void**)&dev_gridParticleIndex, sizeof(unsigned int)*np);
    hipMalloc((void**)&dev_cellStart, sizeof(unsigned int)
               *grid.num[0]*grid.num[1]*grid.num[2]);
    hipMalloc((void**)&dev_cellEnd, sizeof(unsigned int)
               *grid.num[0]*grid.num[1]*grid.num[2]);

    // Host contact bookkeeping arrays
    k.contacts = new unsigned int[np*NC];
    // Initialize contacts lists to np
#pragma omp parallel for if(np>100)
    for (unsigned int i=0; i<(np*NC); ++i)
        k.contacts[i] = np;
    k.distmod = new Float4[np*NC];
    k.delta_t = new Float4[np*NC];

    // Wall arrays
    hipMalloc((void**)&dev_walls_wmode, sizeof(int)*walls.nw);
    hipMalloc((void**)&dev_walls_nx, sizeof(Float4)*walls.nw);
    hipMalloc((void**)&dev_walls_mvfd, sizeof(Float4)*walls.nw);
    hipMalloc((void**)&dev_walls_force_pp, sizeof(Float)*walls.nw*np);
    hipMalloc((void**)&dev_walls_acc, sizeof(Float)*walls.nw);
    // dev_walls_force_partial allocated later

    checkForCudaErrors("End of allocateGlobalDeviceMemory");
    if (verbose == 1)
        std::cout << "Done" << std::endl;
}

// Allocate global memory on other devices required for "interact" function.
// The values of domain_size[ndevices] must be set beforehand.
__host__ void DEM::allocateHelperDeviceMemory(void)
{
    // Particle memory size
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Initialize pointers to per-GPU arrays
    hdev_gridParticleIndex = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_cellStart         = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_cellEnd           = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_x                 = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_x_sorted          = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_vel               = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_vel_sorted        = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_angvel            = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_angvel_sorted     = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_walls_nx          = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_walls_mvfd        = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_distmod           = (Float4**)malloc(ndevices*sizeof(Float4*));

    hdev_force_sorted          = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_torque_sorted         = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_delta_t_sorted        = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_es_dot_sorted         = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_ev_dot_sorted         = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_p_sorted              = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_walls_force_pp_sorted = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_contacts_sorted       = (unsigned**)malloc(ndevices*sizeof(unsigned*));

    for (int d=0; d<ndevices; d++) {

        // do not allocate memory on primary GPU
        if (d == device)
            continue;

        hipSetDevice(d);

        // allocate space for full input arrays for interact()
        hipMalloc((void**)&hdev_gridParticleIndex[d], sizeof(unsigned int)*np);
        hipMalloc((void**)&hdev_cellStart[d], sizeof(unsigned int)
                   *grid.num[0]*grid.num[1]*grid.num[2]);
        hipMalloc((void**)&hdev_cellEnd[d], sizeof(unsigned int)
                   *grid.num[0]*grid.num[1]*grid.num[2]);
        hipMalloc((void**)&hdev_x[d], memSizeF4);
        hipMalloc((void**)&hdev_x_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_vel[d], memSizeF4);
        hipMalloc((void**)&hdev_vel_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_angvel[d], memSizeF4);
        hipMalloc((void**)&hdev_angvel_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_walls_nx[d], sizeof(Float4)*walls.nw);
        hipMalloc((void**)&hdev_walls_mvfd[d], sizeof(Float4)*walls.nw);
        hipMalloc((void**)&hdev_distmod[d], memSizeF4*NC);

        // allocate space for partial output arrays for interact()
        hipMalloc((void**)&hdev_force_sorted[d], sizeof(Float4)*domain_size[d]);
        hipMalloc((void**)&hdev_torque_sorted[d], sizeof(Float4)*domain_size[d]);
        hipMalloc((void**)&hdev_es_dot_sorted[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_ev_dot_sorted[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_p_sorted[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_walls_force_pp_sorted[d],
                   sizeof(Float)*domain_size[d]*walls.nw);
        hipMalloc((void**)&hdev_contacts_sorted[d],
                   sizeof(unsigned)*domain_size[d]*NC);
        hipMalloc((void**)&hdev_delta_t_sorted[d],
                   sizeof(Float4)*domain_size[d]*NC);

        checkForCudaErrors("During allocateGlobalDeviceMemoryOtherDevices");
    }
    hipSetDevice(device); // select main device
}

// Create streams for asynchronous operations
__host__ void DEM::createHelperStreams()
{
    // streams for asynchronous command execution
    stream = (hipStream_t*)malloc(sizeof(hipStream_t)*ndevices);
    
    for (int d=0; d<ndevices; d++) {
        hipSetDevice(d);
        hipStreamCreate(&stream[d]);
        checkForCudaErrors("During createHelperStreams");
    }
}

__host__ void DEM::destroyHelperStreams()
{
    for (int d=0; d<ndevices; d++) {
        hipSetDevice(d);
        hipStreamDestroy(stream[d]);
        checkForCudaErrors("During createHelperStreams");
    }
    free(stream);
}

// Transfer full input array values from main devices to helper devices.
// Function could be accelerated by asynchronous memory transfers
// (hipMemcpyPeerAsync), which require streams.
__host__ void DEM::transferToHelperDevices()
{
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // from main device to host
    //transferFromGlobalDeviceMemory();
    
    for (int d=0; d<ndevices; d++) {

        if (d == device)
            continue;

        hipSetDevice(d);

        // copy all input memory from main device to helper device(s)
        hipMemcpyPeerAsync(hdev_gridParticleIndex[d], d,
                            dev_gridParticleIndex, device,
                            sizeof(unsigned)*np, stream[d]);
        hipMemcpyPeerAsync(hdev_cellStart[d], d,
                            dev_cellStart, device,
                            sizeof(unsigned)*grid.num[0]*grid.num[1]*grid.num[2],
                            stream[d]);
        hipMemcpyPeerAsync(hdev_cellEnd[d], d,
                            dev_cellEnd, device,
                            sizeof(unsigned)*grid.num[0]*grid.num[1]*grid.num[2],
                            stream[d]);
        hipMemcpyPeerAsync(hdev_x[d], d, dev_x, device, memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_x_sorted[d], d, dev_x_sorted, device,
                            memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_vel[d], d, dev_vel, device,
                            memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_vel_sorted[d], d, dev_vel_sorted, device,
                            memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_angvel[d], d, dev_angvel, device,
                            memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_angvel_sorted[d], d, dev_angvel_sorted, device,
                            memSizeF4, stream[d]);
        hipMemcpyPeerAsync(hdev_walls_nx[d], d, dev_walls_nx, device,
                            sizeof(Float4)*walls.nw, stream[d]);
        hipMemcpyPeerAsync(hdev_walls_mvfd[d], d, dev_walls_mvfd, device,
                            sizeof(Float4)*walls.nw, stream[d]);
        hipMemcpyPeerAsync(hdev_distmod[d], d, dev_distmod, device,
                            memSizeF4*NC, stream[d]);

        // TODO: copy energy sum arrays due to += operations in interact() or
        // create a separate kernel which does es += es_dot * devC_dt

    }

    for (int d=0; d<ndevices; d++) {
        if (d == device)
            continue;
        hipSetDevice(d);
        checkForCudaErrors("During transferToHelperDevice");
        hipStreamSynchronize(stream[d]);
    }
    hipSetDevice(device); // select main device
}

// Transfer piecewise output array values from helper devices to main device
__host__ void DEM::transferFromHelperDevices()
{
    for (int d=0; d<ndevices; d++) {

        if (d == device)
            continue;

        hipSetDevice(d);


    }
}



__host__ void DEM::freeHelperDeviceMemory()
{
    for (int d=0; d<ndevices; d++) {

        // do not allocate memory on primary GPU
        if (d == device)
            continue;

        hipSetDevice(d);

        hipFree(hdev_gridParticleIndex[d]);
        hipFree(hdev_cellStart[d]);
        hipFree(hdev_cellEnd[d]);
        hipFree(hdev_x[d]);
        hipFree(hdev_vel[d]);
        hipFree(hdev_vel_sorted[d]);
        hipFree(hdev_angvel[d]);
        hipFree(hdev_angvel_sorted[d]);
        hipFree(hdev_walls_nx[d]);
        hipFree(hdev_walls_mvfd[d]);
        hipFree(hdev_distmod[d]);

        hipFree(hdev_force_sorted[d]);
        hipFree(hdev_torque_sorted[d]);
        hipFree(hdev_es_dot_sorted[d]);
        hipFree(hdev_ev_dot_sorted[d]);
        hipFree(hdev_p_sorted[d]);
        hipFree(hdev_walls_force_pp_sorted[d]);
        hipFree(hdev_contacts_sorted[d]);
        hipFree(hdev_delta_t_sorted[d]);

        checkForCudaErrors("During helper device hipFree calls");
    }
    hipSetDevice(device); // select primary GPU
}

__host__ void DEM::freeGlobalDeviceMemory()
{
    if (verbose == 1)
        printf("\nFreeing device memory:                           ");

    // Particle arrays
    hipFree(dev_x);
    hipFree(dev_xyzsum);
    hipFree(dev_vel);
    hipFree(dev_vel0);
    hipFree(dev_acc);
    hipFree(dev_force);
    hipFree(dev_angpos);
    hipFree(dev_angvel);
    hipFree(dev_angvel0);
    hipFree(dev_angacc);
    hipFree(dev_torque);

    hipFree(dev_contacts);
    hipFree(dev_distmod);
    hipFree(dev_delta_t);
    hipFree(dev_bonds);
    hipFree(dev_bonds_delta);
    hipFree(dev_bonds_omega);

    hipFree(dev_es_dot);
    hipFree(dev_es);
    hipFree(dev_ev_dot);
    hipFree(dev_ev);
    hipFree(dev_p);

    hipFree(dev_x_sorted);
    hipFree(dev_vel_sorted);
    hipFree(dev_angvel_sorted);

    // Cell-related arrays
    hipFree(dev_gridParticleIndex);
    hipFree(dev_cellStart);
    hipFree(dev_cellEnd);

    // Wall arrays
    hipFree(dev_walls_nx);
    hipFree(dev_walls_mvfd);
    hipFree(dev_walls_force_partial);
    hipFree(dev_walls_force_pp);
    hipFree(dev_walls_acc);

    // Fluid arrays
    if (navierstokes == 1) {
        freeNSmemDev();
    }

    //checkForCudaErrors("During hipFree calls");

    if (verbose == 1)
        std::cout << "Done" << std::endl;
}


__host__ void DEM::transferToGlobalDeviceMemory(int statusmsg)
{
    if (verbose == 1 && statusmsg == 1)
        std::cout << "  Transfering data to the device:                 ";

    // Commonly-used memory sizes
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Copy static-size structure data from host to global device memory
    //hipMemcpy(dev_time, &time, sizeof(Time), hipMemcpyHostToDevice);

    // Kinematic particle values
    hipMemcpy( dev_x,	       k.x,	   
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_xyzsum,    k.xyzsum,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_vel,      k.vel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_vel0,     k.vel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_acc,      k.acc, 
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_force,    k.force,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angpos,   k.angpos,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angvel,   k.angvel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angvel0,  k.angvel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angacc,   k.angacc,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_torque,   k.torque,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_contacts, k.contacts,
                sizeof(unsigned int)*np*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_distmod, k.distmod,
                memSizeF4*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_delta_t, k.delta_t,
                memSizeF4*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds, k.bonds,
                sizeof(uint2)*params.nb0, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds_delta, k.bonds_delta,
                sizeof(Float4)*params.nb0, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds_omega, k.bonds_omega,
                sizeof(Float4)*params.nb0, hipMemcpyHostToDevice);

    // Individual particle energy values
    hipMemcpy( dev_es_dot, e.es_dot,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_es,     e.es,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_ev_dot, e.ev_dot,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_ev,     e.ev,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_p, e.p,
                memSizeF, hipMemcpyHostToDevice);

    // Wall parameters
    hipMemcpy( dev_walls_wmode, walls.wmode,
                sizeof(int)*walls.nw, hipMemcpyHostToDevice);
    hipMemcpy( dev_walls_nx,    walls.nx,
                sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);
    hipMemcpy( dev_walls_mvfd,  walls.mvfd,
                sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);

    // Fluid arrays
    if (navierstokes == 1) {
        transferNStoGlobalDeviceMemory(1);
    } else if (navierstokes != 0) {
        std::cerr << "Error: navierstokes value not understood ("
            << navierstokes << ")" << std::endl;
    }

    checkForCudaErrors("End of transferToGlobalDeviceMemory");
    if (verbose == 1 && statusmsg == 1)
        std::cout << "Done" << std::endl;
}

__host__ void DEM::transferFromGlobalDeviceMemory()
{
    //std::cout << "  Transfering data from the device:               ";

    // Commonly-used memory sizes
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Copy static-size structure data from host to global device memory
    //hipMemcpy(&time, dev_time, sizeof(Time), hipMemcpyDeviceToHost);

    // Kinematic particle values
    hipMemcpy( k.x, dev_x,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.xyzsum, dev_xyzsum,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.vel, dev_vel,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.acc, dev_acc,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.force, dev_force,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angpos, dev_angpos,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angvel, dev_angvel,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angacc, dev_angacc,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.torque, dev_torque,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.contacts, dev_contacts,
            sizeof(unsigned int)*np*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.distmod, dev_distmod,
            memSizeF4*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.delta_t, dev_delta_t,
            memSizeF4*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds, dev_bonds,
            sizeof(uint2)*params.nb0, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds_delta, dev_bonds_delta,
            sizeof(Float4)*params.nb0, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds_omega, dev_bonds_omega,
            sizeof(Float4)*params.nb0, hipMemcpyDeviceToHost);

    // Individual particle energy values
    hipMemcpy( e.es_dot, dev_es_dot,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.es, dev_es,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.ev_dot, dev_ev_dot,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.ev, dev_ev,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.p, dev_p,
            memSizeF, hipMemcpyDeviceToHost);

    // Wall parameters
    hipMemcpy( walls.wmode, dev_walls_wmode,
            sizeof(int)*walls.nw, hipMemcpyDeviceToHost);
    hipMemcpy( walls.nx, dev_walls_nx,
            sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);
    hipMemcpy( walls.mvfd, dev_walls_mvfd,
            sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);

    // Fluid arrays
    if (navierstokes == 1) {
        transferNSfromGlobalDeviceMemory(0);
    }

    //checkForCudaErrors("End of transferFromGlobalDeviceMemory");
}


// Iterate through time by explicit time integration
__host__ void DEM::startTime()
{
    using std::cout;
    using std::cerr;
    using std::endl;

    std::string outfile;
    char file[200];
    FILE *fp;

    // Synchronization point
    hipDeviceSynchronize();
    checkForCudaErrors("Start of startTime()");

    // Write initial data to output/<sid>.output00000.bin
    writebin(("output/" + sid + ".output00000.bin").c_str());

    // Time variables
    clock_t tic, toc;
    double filetimeclock, time_spent;
    float dev_time_spent;

    // Start CPU clock
    tic = clock();

    //// GPU workload configuration
    unsigned int threadsPerBlock = 256; 
    //unsigned int threadsPerBlock = 512; 

    // Create enough blocks to accomodate the particles
    unsigned int blocksPerGrid = iDivUp(np, threadsPerBlock); 
    dim3 dimGrid(blocksPerGrid, 1, 1); // Blocks arranged in 1D grid
    dim3 dimBlock(threadsPerBlock, 1, 1); // Threads arranged in 1D block

    unsigned int blocksPerGridBonds = iDivUp(params.nb0, threadsPerBlock); 
    dim3 dimGridBonds(blocksPerGridBonds, 1, 1); // Blocks arranged in 1D grid

    // Use 3D block and grid layout for cell-centered fluid calculations
    dim3 dimBlockFluid(8, 8, 8);    // 512 threads per block
    dim3 dimGridFluid(
            iDivUp(grid.num[0], dimBlockFluid.x),
            iDivUp(grid.num[1], dimBlockFluid.y),
            iDivUp(grid.num[2], dimBlockFluid.z));
    if (dimGridFluid.z > 64 && navierstokes == 1) {
        cerr << "Error: dimGridFluid.z > 64" << endl;
        exit(1);
    }

    // Use 3D block and grid layout for cell-face fluid calculations
    dim3 dimBlockFluidFace(8, 8, 8);    // 512 threads per block
    dim3 dimGridFluidFace(
            iDivUp(grid.num[0]+1, dimBlockFluidFace.x),
            iDivUp(grid.num[1]+1, dimBlockFluidFace.y),
            iDivUp(grid.num[2]+1, dimBlockFluidFace.z));
    if (dimGridFluidFace.z > 64 && navierstokes == 1) {
        cerr << "Error: dimGridFluidFace.z > 64" << endl;
        exit(1);
    }


    // Shared memory per block
    unsigned int smemSize = sizeof(unsigned int)*(threadsPerBlock+1);

    // Pre-sum of force per wall
    hipMalloc((void**)&dev_walls_force_partial,
            sizeof(Float)*dimGrid.x*walls.nw);

    // Report to stdout
    if (verbose == 1) {
        cout << "\n  Device memory allocation and transfer complete.\n"
            << "  - Blocks per grid: "
            << dimGrid.x << "*" << dimGrid.y << "*" << dimGrid.z << "\n"
            << "  - Threads per block: "
            << dimBlock.x << "*" << dimBlock.y << "*" << dimBlock.z << "\n"
            << "  - Shared memory required per block: " << smemSize << " bytes"
            << endl;
        if (navierstokes == 1) {
            cout << "  - Blocks per fluid grid: "
                << dimGridFluid.x << "*" << dimGridFluid.y << "*" <<
                dimGridFluid.z << "\n"
                << "  - Threads per fluid block: "
                << dimBlockFluid.x << "*" << dimBlockFluid.y << "*" <<
                dimBlockFluid.z << endl;
        }
    }

    // Initialize counter variable values
    filetimeclock = 0.0;
    long iter = 0;
    const int stdout_report = 10; // no of steps between reporting to stdout

    // Create first status.dat
    //sprintf(file,"output/%s.status.dat", sid);
    outfile = "output/" + sid + ".status.dat";
    fp = fopen(outfile.c_str(), "w");
    fprintf(fp,"%2.4e %2.4e %d\n", 
            time.current, 
            100.0*time.current/time.total, 
            time.step_count);
    fclose(fp);

    if (verbose == 1) {
        cout << "\n  Entering the main calculation time loop...\n\n"
            << "  IMPORTANT: Do not close this terminal, doing so will \n"
            << "             terminate this SPHERE process. Follow the \n"
            << "             progress by executing:\n"
            << "                $ ./sphere_status " << sid << endl << endl;
    }


    // Start GPU clock
    hipEvent_t dev_tic, dev_toc;
    hipEventCreate(&dev_tic);
    hipEventCreate(&dev_toc);
    hipEventRecord(dev_tic, 0);

    // If profiling is enabled, initialize timers for each kernel
    hipEvent_t kernel_tic, kernel_toc;
    float kernel_elapsed;
    double t_calcParticleCellID = 0.0;
    double t_thrustsort = 0.0;
    double t_reorderArrays = 0.0;
    double t_topology = 0.0;
    double t_interact = 0.0;
    double t_bondsLinear = 0.0;
    double t_latticeBoltzmannD3Q19 = 0.0;
    double t_integrate = 0.0;
    double t_summation = 0.0;
    double t_integrateWalls = 0.0;

    double t_findPorositiesDev = 0.0;
    double t_findNSstressTensor = 0.0;
    double t_findNSdivphiviv = 0.0;
    double t_findNSdivtau = 0.0;
    double t_findPredNSvelocities = 0.0;
    double t_setNSepsilon = 0.0;
    double t_setNSdirichlet = 0.0;
    double t_setNSghostNodesDev = 0.0;
    double t_findNSforcing = 0.0;
    double t_jacobiIterationNS = 0.0;
    double t_updateNSvelocityPressure = 0.0;

    if (PROFILING == 1) {
        hipEventCreate(&kernel_tic);
        hipEventCreate(&kernel_toc);
    }

    // The model start time is saved for profiling performance
    double t_start = time.current;
    double t_ratio;     // ration between time flow in model vs. reality

    // Write a log file of the number of iterations it took before
    // convergence in the fluid solver
    std::ofstream convlog;
    if (write_conv_log == 1) {
        std::string f = "output/" + sid + "-conv.log";
        convlog.open(f.c_str());
    }

    if (verbose == 1)
        cout << "  Current simulation time: " << time.current << " s.";


    // MAIN CALCULATION TIME LOOP
    while (time.current <= time.total) {

        // Print current step number to terminal
        //printf("\n\n@@@ DEM time step: %ld\n", iter);

        // Routine check for errors
        checkForCudaErrors("Start of main while loop");

        if (np > 0) {

            // For each particle: 
            // Compute hash key (cell index) from position 
            // in the fine, uniform and homogenous grid.
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            calcParticleCellID<<<dimGrid, dimBlock>>>(dev_gridParticleCellID,
                    dev_gridParticleIndex, 
                    dev_x);

            // Synchronization point
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_calcParticleCellID);
            checkForCudaErrorsIter("Post calcParticleCellID", iter);


            // Sort particle (key, particle ID) pairs by hash key with Thrust
            // radix sort
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            thrust::sort_by_key(
                    thrust::device_ptr<uint>(dev_gridParticleCellID),
                    thrust::device_ptr<uint>(dev_gridParticleCellID + np),
                    thrust::device_ptr<uint>(dev_gridParticleIndex));
            hipDeviceSynchronize(); // Maybe Thrust synchronizes implicitly?
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_thrustsort);
            checkForCudaErrorsIter("Post thrust::sort_by_key", iter);


            // Zero cell array values by setting cellStart to its highest
            // possible value, specified with pointer value 0xffffffff, which
            // for a 32 bit unsigned int is 4294967295.
            hipMemset(dev_cellStart, 0xffffffff, 
                    grid.num[0]*grid.num[1]*grid.num[2]*sizeof(unsigned int));
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post hipMemset", iter);

            // Use sorted order to reorder particle arrays (position,
            // velocities, radii) to ensure coherent memory access. Save ordered
            // configurations in new arrays (*_sorted).
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            reorderArrays<<<dimGrid, dimBlock, smemSize>>>(dev_cellStart, 
                    dev_cellEnd,
                    dev_gridParticleCellID, 
                    dev_gridParticleIndex,
                    dev_x, dev_vel, 
                    dev_angvel,
                    dev_x_sorted, 
                    dev_vel_sorted, 
                    dev_angvel_sorted);

            // Synchronization point
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_reorderArrays);
            checkForCudaErrorsIter("Post reorderArrays", iter);

            // The contact search in topology() is only necessary for
            // determining the accumulated shear distance needed in the linear
            // elastic and nonlinear contact force model
            if (params.contactmodel == 2 || params.contactmodel == 3) {
                // For each particle: Search contacts in neighbor cells
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                topology<<<dimGrid, dimBlock>>>(dev_cellStart, 
                        dev_cellEnd,
                        dev_gridParticleIndex,
                        dev_x_sorted, 
                        dev_contacts,
                        dev_distmod);

                // Synchronization point
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_topology);
                checkForCudaErrorsIter(
                        "Post topology: One or more particles moved "
                        "outside the grid.\nThis could possibly be caused by a "
                        "numerical instability.\nIs the computational time step"
                        " too large?", iter);
            }

            // For each particle process collisions and compute resulting forces
            //cudaPrintfInit();
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            interact<<<dimGrid, dimBlock>>>(dev_gridParticleIndex,
                    dev_cellStart,
                    dev_cellEnd,
                    dev_x,
                    dev_x_sorted,
                    dev_vel_sorted,
                    dev_angvel_sorted,
                    dev_vel,
                    dev_angvel,
                    dev_force, 
                    dev_torque, 
                    dev_es_dot,
                    dev_ev_dot, 
                    dev_es,
                    dev_ev,
                    dev_p,
                    dev_walls_nx,
                    dev_walls_mvfd,
                    dev_walls_force_pp,
                    dev_contacts,
                    dev_distmod,
                    dev_delta_t);

            // Synchronization point
            hipDeviceSynchronize();
            //cudaPrintfDisplay(stdout, true);
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_interact);
            checkForCudaErrorsIter(
                    "Post interact - often caused if particles move "
                    "outside the grid", iter);

            // Process particle pairs
            if (params.nb0 > 0) {
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                bondsLinear<<<dimGridBonds, dimBlock>>>(
                        dev_bonds,
                        dev_bonds_delta,
                        dev_bonds_omega,
                        dev_x,
                        dev_vel,
                        dev_angvel,
                        dev_force,
                        dev_torque);
                // Synchronization point
                hipDeviceSynchronize();
                //cudaPrintfDisplay(stdout, true);
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_bondsLinear);
                checkForCudaErrorsIter("Post bondsLinear", iter);
            }
        }

        // Solve Navier Stokes flow through the grid
        if (navierstokes == 1) {
            checkForCudaErrorsIter("Before findPorositiesDev", iter);
            // Find cell porosities, average particle velocities, and average
            // particle diameters. These are needed for predicting the fluid
            // velocities
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            findPorositiesVelocitiesDiametersSpherical
            //findPorositiesVelocitiesDiametersSphericalGradient
                <<<dimGridFluid, dimBlockFluid>>>(
                        dev_cellStart,
                        dev_cellEnd,
                        dev_x_sorted,
                        dev_vel_sorted,
                        dev_ns_phi,
                        dev_ns_dphi,
                        dev_ns_vp_avg,
                        dev_ns_d_avg,
                        iter,
                        np);
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_findPorositiesDev);
            checkForCudaErrorsIter("Post findPorositiesDev", iter);

#ifdef CFD_DEM_COUPLING
            /*if (params.nu <= 0.0) {
                std::cerr << "Error! The fluid needs a positive viscosity "
                    "value in order to simulate particle-fluid interaction."
                    << std::endl;
                exit(1);
            }*/
            if (iter == 0) {
                // set cell center ghost nodes
                setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                    dev_ns_v, ns.bc_bot, ns.bc_top);

                // find cell face velocities
                interpolateCenterToFace
                    <<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_v,
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z);
                hipDeviceSynchronize();
                checkForCudaErrors("Post interpolateCenterToFace");
            }

            setNSghostNodesFace<Float>
                <<<dimGridFluidFace, dimBlockFluidFace>>>(
                    dev_ns_v_x,
                    dev_ns_v_y,
                    dev_ns_v_z,
                    ns.bc_bot,
                    ns.bc_top);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post setNSghostNodesFace", iter);

            findFaceDivTau<<<dimGridFluidFace, dimBlockFluidFace>>>(
                dev_ns_v_x,
                dev_ns_v_y,
                dev_ns_v_z,
                dev_ns_div_tau_x,
                dev_ns_div_tau_y,
                dev_ns_div_tau_z);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post findFaceDivTau", iter);

            setNSghostNodesFace<Float>
                <<<dimGridFluidFace, dimBlockFluid>>>(
                    dev_ns_div_tau_x,
                    dev_ns_div_tau_y,
                    dev_ns_div_tau_z,
                    ns.bc_bot,
                    ns.bc_top);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_div_tau)",
                                   iter);

            setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                dev_ns_p, ns.bc_bot, ns.bc_top);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)", iter);

            setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                dev_ns_phi, ns.bc_bot, ns.bc_top);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)", iter);


            if (np > 0) {

                // Per particle, find the fluid-particle interaction force f_pf
                // and apply it to the particle
                findInteractionForce<<<dimGrid, dimBlock>>>(
                        dev_x,
                        dev_vel,
                        dev_ns_phi,
                        dev_ns_p,
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        dev_ns_div_tau_x,
                        dev_ns_div_tau_y,
                        dev_ns_div_tau_z,
                        dev_ns_f_pf,
                        dev_force);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post findInteractionForce", iter);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_p, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)", iter);

                // Apply fluid-particle interaction force to the fluid
                applyInteractionForceToFluid<<<dimGridFluid, dimBlockFluid>>>(
                        dev_gridParticleIndex,
                        dev_cellStart,
                        dev_cellEnd,
                        dev_ns_f_pf,
                        dev_ns_F_pf);
                        //dev_ns_F_pf_x,
                        //dev_ns_F_pf_y,
                        //dev_ns_F_pf_z);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post applyInteractionForceToFluid",
                        iter);

                setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_F_pf, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(F_pf)", iter);
            }
#endif

            if ((iter % ns.ndem) == 0) {
                // Initial guess for the top epsilon values. These may be
                // changed in setUpperPressureNS
                Float pressure = ns.p[idx(0,0,ns.nz-1)];
                Float pressure_new = pressure; // Dirichlet
                Float epsilon_value = pressure_new - ns.beta*pressure;
                setNSepsilonTop<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_epsilon,
                        dev_ns_epsilon_new,
                        epsilon_value);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSepsilonTop", iter);

                // Modulate the pressures at the upper boundary cells
                if ((ns.p_mod_A > 1.0e-5 || ns.p_mod_A < -1.0e-5) &&
                        ns.p_mod_f > 1.0e-7) {
                                         // original pressure
                    Float new_pressure = ns.p[idx(0,0,ns.nz-1)]
                        + ns.p_mod_A*sin(2.0*M_PI*ns.p_mod_f*time.current
                                + ns.p_mod_phi);
                    setUpperPressureNS<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_p,
                            dev_ns_epsilon,
                            dev_ns_epsilon_new,
                            ns.beta,
                            new_pressure);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setUpperPressureNS", iter);

#ifdef REPORT_MORE_EPSILON
                    std::cout
                        << "\n@@@@@@ TIME STEP " << iter << " @@@@@@"
                        << "\n###### EPSILON AFTER setUpperPressureNS "
                        << "######" << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif
                }

                // Set the values of the ghost nodes in the grid
                if (PROFILING == 1)
                    startTimer(&kernel_tic);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_p, ns.bc_bot, ns.bc_top);

                //setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                        //dev_ns_v, ns.bc_bot, ns.bc_top);

                setNSghostNodesFace<Float>
                    <<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_v_p_x,
                        dev_ns_v_p_y,
                        dev_ns_v_p_z,
                        ns.bc_bot, ns.bc_top);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_phi, ns.bc_bot, ns.bc_top);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_dphi, ns.bc_bot, ns.bc_top);

                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_setNSghostNodesDev);
                checkForCudaErrorsIter("Post setNSghostNodesDev", iter);
                /*std::cout << "\n###### EPSILON AFTER setNSghostNodesDev #####"
                  << std::endl;
                  transferNSepsilonFromGlobalDeviceMemory();
                  printNSarray(stdout, ns.epsilon, "epsilon");*/

                // interpolate velocities to cell centers which makes velocity
                // prediction easier
                interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        dev_ns_v);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post interpolateFaceToCenter", iter);

                // Set cell-center velocity ghost nodes
                setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                    dev_ns_v, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(v)", iter);
                
                // Find the divergence of phi*vi*v, needed for predicting the
                // fluid velocities
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                findNSdivphiviv<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_phi,
                        dev_ns_v,
                        dev_ns_div_phi_vi_v);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_findNSdivphiviv);
                checkForCudaErrorsIter("Post findNSdivphiviv", iter);

                // Set cell-center ghost nodes
                setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                    dev_ns_div_phi_vi_v, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(div_phi_vi_v)",
                                       iter);

                // Predict the fluid velocities on the base of the old pressure
                // field and ignoring the incompressibility constraint
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                findPredNSvelocities<<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_p,
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        dev_ns_phi,
                        dev_ns_dphi,
                        dev_ns_div_tau_x,
                        dev_ns_div_tau_y,
                        dev_ns_div_tau_z,
                        dev_ns_div_phi_vi_v,
                        ns.bc_bot,
                        ns.bc_top,
                        ns.beta,
                        dev_ns_F_pf,
                        ns.ndem,
                        dev_ns_v_p_x,
                        dev_ns_v_p_y,
                        dev_ns_v_p_z);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_findPredNSvelocities);
                checkForCudaErrorsIter("Post findPredNSvelocities", iter);

                setNSghostNodesFace<Float>
                    <<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_v_p_x,
                        dev_ns_v_p_y,
                        dev_ns_v_p_z,
                        ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter(
                        "Post setNSghostNodesFace(dev_ns_v_p)", iter);

                interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_v_p_x,
                        dev_ns_v_p_y,
                        dev_ns_v_p_z,
                        dev_ns_v_p);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post interpolateFaceToCenter", iter);


                // In the first iteration of the sphere program, we'll need to
                // manually estimate the values of epsilon. In the subsequent
                // iterations, the previous values are  used.
                if (iter == 0) {

                    // Define the first estimate of the values of epsilon.
                    // The initial guess depends on the value of ns.beta.
                    Float pressure = ns.p[idx(2,2,2)];
                    Float pressure_new = pressure; // Guess p_current = p_new
                    Float epsilon_value = pressure_new - ns.beta*pressure;
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setNSepsilonInterior<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon, epsilon_value);
                    hipDeviceSynchronize();

                    setNSnormZero<<<dimGridFluid, dimBlockFluid>>>(dev_ns_norm);
                    hipDeviceSynchronize();

                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setNSepsilon);
                    checkForCudaErrorsIter("Post setNSepsilonInterior", iter);

#ifdef REPORT_MORE_EPSILON
                    std::cout
                        << "\n###### EPSILON AFTER setNSepsilonInterior "
                        << "######" << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                    // Set the epsilon values at the lower boundary
                    pressure = ns.p[idx(0,0,0)];
                    pressure_new = pressure; // Guess p_current = p_new
                    epsilon_value = pressure_new - ns.beta*pressure;
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setNSepsilonBottom<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            dev_ns_epsilon_new,
                            epsilon_value);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setNSdirichlet);
                    checkForCudaErrorsIter("Post setNSepsilonBottom", iter);

#ifdef REPORT_MORE_EPSILON
                    std::cout
                        << "\n###### EPSILON AFTER setNSepsilonBottom "
                        << "######" << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                    /*setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_epsilon);
                      hipDeviceSynchronize();
                      checkForCudaErrors("Post setNSghostNodesFloat(dev_ns_epsilon)",
                      iter);*/
                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodesEpsilon(1)",
                            iter);

#ifdef REPORT_MORE_EPSILON
                    std::cout <<
                        "\n###### EPSILON AFTER setNSghostNodes(epsilon) "
                        << "######" << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif
                }

                // Solve the system of epsilon using a Jacobi iterative solver.
                // The average normalized residual is initialized to a large
                // value.
                //double avg_norm_res;
                double max_norm_res;

                // Write a log file of the normalized residuals during the Jacobi
                // iterations
                std::ofstream reslog;
                if (write_res_log == 1)
                    reslog.open("max_res_norm.dat");

                // transfer normalized residuals from GPU to CPU
#ifdef REPORT_MORE_EPSILON
                std::cout << "\n###### BEFORE FIRST JACOBI ITERATION ######"
                    << "\n@@@@@@ TIME STEP " << iter << " @@@@@@"
                    << std::endl;
                transferNSepsilonFromGlobalDeviceMemory();
                printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                for (unsigned int nijac = 0; nijac<ns.maxiter; ++nijac) {

                    //printf("### Jacobi iteration %d\n", nijac);

                    // Only grad(epsilon) changes during the Jacobi iterations.
                    // The remaining terms of the forcing function are only
                    // calculated during the first iteration.
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findNSforcing<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            dev_ns_phi,
                            dev_ns_dphi,
                            dev_ns_v_p,
                            dev_ns_v_p_x,
                            dev_ns_v_p_y,
                            dev_ns_v_p_z,
                            nijac,
                            ns.ndem,
                            dev_ns_f1,
                            dev_ns_f2,
                            dev_ns_f);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findNSforcing);
                    checkForCudaErrorsIter("Post findNSforcing", iter);
                    /*setNSghostNodesForcing<<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_f1,
                      dev_ns_f2,
                      dev_ns_f,
                      nijac);
                      hipDeviceSynchronize();
                      checkForCudaErrors("Post setNSghostNodesForcing", iter);*/

                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodesEpsilon(2)",
                            iter);

#ifdef REPORT_EPSILON
                    std::cout << "\n###### JACOBI ITERATION "
                        << nijac
                        << " after setNSghostNodes(epsilon,2) ######"
                        << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                    /*smoothing<Float><<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_epsilon,
                      ns.bc_bot, ns.bc_top);
                      hipDeviceSynchronize();
                      checkForCudaErrorsIter("Post smoothing", iter);

#ifdef REPORT_EPSILON
                      std::cout << "\n###### JACOBI ITERATION "
                      << nijac << " after smoothing(epsilon) ######"
                      << std::endl;
                      transferNSepsilonFromGlobalDeviceMemory();
                      printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                      setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_epsilon,
                      ns.bc_bot, ns.bc_top);
                      hipDeviceSynchronize();
                      checkForCudaErrorsIter("Post setNSghostNodesEpsilon(3)",
                      iter);
                     */

                    /*if (report_epsilon == 1) {
                      std::cout << "\n###### JACOBI ITERATION "
                      << nijac
                      << " after setNSghostNodesEpsilon(epsilon,3) ######"
                      << std::endl;
                      transferNSepsilonFromGlobalDeviceMemory();
                      printNSarray(stdout, ns.epsilon, "epsilon");
                      }*/

                    // Store old values
                    /*copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_epsilon,
                      dev_ns_epsilon_old);
                      hipDeviceSynchronize();
                      checkForCudaErrorsIter
                          ("Post copyValues (epsilon->epsilon_old)", iter);*/

                    // Perform a single Jacobi iteration
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    jacobiIterationNS<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            dev_ns_epsilon_new,
                            dev_ns_norm,
                            dev_ns_f,
                            ns.bc_bot,
                            ns.bc_top,
                            ns.theta);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_jacobiIterationNS);
                    checkForCudaErrorsIter("Post jacobiIterationNS", iter);

                    // Flip flop: swap new and current array pointers
                    /*Float* tmp         = dev_ns_epsilon;
                      dev_ns_epsilon     = dev_ns_epsilon_new;
                      dev_ns_epsilon_new = tmp;*/

                    // Copy new values to current values
                    copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon_new,
                            dev_ns_epsilon);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter
                        ("Post copyValues (epsilon_new->epsilon)", iter);

                    /*findNormalizedResiduals<<<dimGridFluid, dimBlockFluid>>>(
                      dev_ns_epsilon_old,
                      dev_ns_epsilon,
                      dev_ns_norm,
                      ns.bc_bot, ns.bc_top);
                      hipDeviceSynchronize();
                      checkForCudaErrorsIter("Post findNormalizedResiduals",
                      iter);*/

#ifdef REPORT_EPSILON
                    std::cout << "\n###### JACOBI ITERATION "
                        << nijac << " after jacobiIterationNS ######"
                        << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                    if (nijac % nijacnorm == 0) {

                        // Read the normalized residuals from the device
                        transferNSnormFromGlobalDeviceMemory();

                        // Write the normalized residuals to the terminal
                        //printNSarray(stdout, ns.norm, "norm");

                        // Find the maximum value of the normalized residuals
                        max_norm_res = maxNormResNS();

                        // Write the Jacobi iteration number and maximum value
                        // of the normalized residual to the log file
                        if (write_res_log == 1)
                            reslog << nijac << '\t' << max_norm_res
                                << std::endl;
                    }

                    if (max_norm_res < ns.tolerance) {

                        if (write_conv_log == 1 && iter % conv_log_interval == 0)
                            convlog << iter << '\t' << nijac << std::endl;

                        // Apply smoothing if requested
                        if (ns.gamma > 0.0) {
                            setNSghostNodes<Float>
                                <<<dimGridFluid, dimBlockFluid>>>(
                                    dev_ns_epsilon,
                                    ns.bc_bot, ns.bc_top);
                            hipDeviceSynchronize();
                            checkForCudaErrorsIter
                                ("Post setNSghostNodesEpsilon(4)", iter);

                            smoothing<<<dimGridFluid, dimBlockFluid>>>(
                                    dev_ns_epsilon,
                                    ns.gamma,
                                    ns.bc_bot, ns.bc_top);
                            hipDeviceSynchronize();
                            checkForCudaErrorsIter("Post smoothing", iter);

                            setNSghostNodes<Float>
                                <<<dimGridFluid, dimBlockFluid>>>(
                                    dev_ns_epsilon,
                                    ns.bc_bot, ns.bc_top);
                            hipDeviceSynchronize();
                            checkForCudaErrorsIter
                                ("Post setNSghostNodesEpsilon(4)", iter);
                        }

#ifdef REPORT_EPSILON
                        std::cout << "\n###### JACOBI ITERATION "
                            << nijac << " after smoothing ######"
                            << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                        break;  // solution has converged, exit Jacobi loop
                    }

                    if (nijac >= ns.maxiter-1) {

                        if (write_conv_log == 1)
                            convlog << iter << '\t' << nijac << std::endl;

                        std::cerr << "\nIteration " << iter << ", time " 
                            << iter*time.dt << " s: "
                            "Error, the epsilon solution in the fluid "
                            "calculations did not converge. Try increasing the "
                            "value of 'ns.maxiter' (" << ns.maxiter
                            << ") or increase 'ns.tolerance' ("
                            << ns.tolerance << ")." << std::endl;
                    }
                    //break; // end after Jacobi first iteration
                } // end Jacobi iteration loop

                if (write_res_log == 1)
                    reslog.close();

                // Find the new pressures and velocities
                if (PROFILING == 1)
                    startTimer(&kernel_tic);

                updateNSpressure<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_epsilon,
                        ns.beta,
                        dev_ns_p);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post updateNSpressure", iter);

                updateNSvelocity<<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_v_p_x,
                        dev_ns_v_p_y,
                        dev_ns_v_p_z,
                        dev_ns_phi,
                        dev_ns_epsilon,
                        ns.beta,
                        ns.bc_bot,
                        ns.bc_top,
                        ns.ndem,
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_updateNSvelocityPressure);
                checkForCudaErrorsIter("Post updateNSvelocity", iter);
            }

            /*std::cout << "\n###### ITERATION "
              << iter << " ######" << std::endl;
              transferNSepsilonFromGlobalDeviceMemory();
              printNSarray(stdout, ns.epsilon, "epsilon");*/
            //transferNSepsilonNewFromGlobalDeviceMemory();
            //printNSarray(stdout, ns.epsilon_new, "epsilon_new");
        }

        if (np > 0) {
            // Update particle kinematics
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            integrate<<<dimGrid, dimBlock>>>(dev_x_sorted, 
                    dev_vel_sorted, 
                    dev_angvel_sorted,
                    dev_x, 
                    dev_vel, 
                    dev_angvel,
                    dev_force,
                    dev_torque, 
                    dev_angpos,
                    dev_acc,
                    dev_angacc,
                    dev_vel0,
                    dev_angvel0,
                    dev_xyzsum,
                    dev_gridParticleIndex,
                    iter);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post integrate", iter);
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_integrate);

            // Summation of forces on wall
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            if (walls.nw > 0) {
                summation<<<dimGrid, dimBlock>>>(dev_walls_force_pp,
                        dev_walls_force_partial);
            }
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_summation);
            checkForCudaErrorsIter("Post wall force summation", iter);

            // Update wall kinematics
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            if (walls.nw > 0) {
                integrateWalls<<< 1, walls.nw>>>(
                        dev_walls_nx,
                        dev_walls_mvfd,
                        dev_walls_wmode,
                        dev_walls_force_partial,
                        dev_walls_acc,
                        blocksPerGrid,
                        time.current,
                        iter);
            }
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_integrateWalls);
            checkForCudaErrorsIter("Post integrateWalls", iter);
        }

        // Update timers and counters
        //time.current  = iter*time.dt;
        time.current  += time.dt;
        filetimeclock += time.dt;
        ++iter;

        // Make sure all preceding tasks are complete
        if (hipDeviceSynchronize() != hipSuccess) {
            cerr << "Error during hipDeviceSynchronize()" << endl;
        }

        // Report time to console
        if (verbose == 1 && (iter % stdout_report == 0)) {

            toc = clock();
            time_spent = (toc - tic)/(CLOCKS_PER_SEC); // real time spent

            // Real time it takes to compute a second of model time
            t_ratio = time_spent/(time.current - t_start);

            cout << "\r  Current simulation time: " 
                << time.current << "/"
                << time.total << " s. ("
                << t_ratio << " s_real/s_sim)       "; // << std::flush;
        }


        // Produce output binary if the time interval 
        // between output files has been reached
        if (filetimeclock >= time.file_dt) {

            // Pause the CPU thread until all CUDA calls previously issued are
            // completed
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Beginning of file output section", iter);

            // v_x, v_y, v_z -> v
            if (navierstokes == 1) {
                interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        dev_ns_v);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post interpolateFaceToCenter", iter);
            }

            //// Copy device data to host memory
            transferFromGlobalDeviceMemory();
            checkForCudaErrorsIter("After transferFromGlobalDeviceMemory()",
                    iter);

            // Pause the CPU thread until all CUDA calls previously issued are
            // completed
            hipDeviceSynchronize();

            // Check the numerical stability of the NS solver
            if (navierstokes == 1)
                checkNSstability();

            // Write binary output file
            time.step_count += 1;
            sprintf(file,"output/%s.output%05d.bin", sid.c_str(),
                    time.step_count); writebin(file);

            /*std::cout << "\n###### OUTPUT FILE " << time.step_count << " ######"
                << std::endl;
            transferNSepsilonFromGlobalDeviceMemory();
            printNSarray(stdout, ns.epsilon, "epsilon");*/

            // Write fluid arrays
            /*if (navierstokes == 1) {
                sprintf(file,"output/%s.ns_phi.output%05d.bin", sid.c_str(),
                    time.step_count);
                writeNSarray(ns.phi, file);
            }*/

            if (CONTACTINFO == 1) {
                // Write contact information to stdout
                cout << "\n\n---------------------------\n"
                    << "t = " << time.current << " s.\n"
                    << "---------------------------\n";

                for (int n = 0; n < np; ++n) {
                    cout << "\n## Particle " << n << " ##\n";

                    cout  << "- contacts:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << "[" << nc << "]=" << k.contacts[nc+NC*n] <<
                            '\n';

                    cout << "\n- delta_t:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << k.delta_t[nc+NC*n].x << '\t'
                            << k.delta_t[nc+NC*n].y << '\t'
                            << k.delta_t[nc+NC*n].z << '\t'
                            << k.delta_t[nc+NC*n].w << '\n';

                    cout << "\n- distmod:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << k.distmod[nc+NC*n].x << '\t'
                            << k.distmod[nc+NC*n].y << '\t'
                            << k.distmod[nc+NC*n].z << '\t'
                            << k.distmod[nc+NC*n].w << '\n';
                }
                cout << '\n';
            }

            // Update status.dat at the interval of filetime 
            outfile = "output/" + sid + ".status.dat";
            fp = fopen(outfile.c_str(), "w");
            fprintf(fp,"%2.4e %2.4e %d\n", 
                    time.current, 
                    100.0*time.current/time.total,
                    time.step_count);
            fclose(fp);

            filetimeclock = 0.0;
        }

        // Uncomment break command to stop after the first iteration
        //break;
    }

    if (write_conv_log == 1)
        convlog.close();


    // Stop clock and display calculation time spent
    toc = clock();
    hipEventRecord(dev_toc, 0);
    hipEventSynchronize(dev_toc);

    time_spent = (toc - tic)/(CLOCKS_PER_SEC);
    hipEventElapsedTime(&dev_time_spent, dev_tic, dev_toc);

    if (verbose == 1) {
        cout << "\nSimulation ended. Statistics:\n"
            << "  - Last output file number: " 
            << time.step_count << "\n"
            << "  - GPU time spent: "
            << dev_time_spent/1000.0f << " s\n"
            << "  - CPU time spent: "
            << time_spent << " s\n"
            << "  - Mean duration of iteration:\n"
            << "      " << dev_time_spent/((double)iter*1000.0f) << " s"
            << std::endl; 
    }

    hipEventDestroy(dev_tic);
    hipEventDestroy(dev_toc);

    hipEventDestroy(kernel_tic);
    hipEventDestroy(kernel_toc);

    // Report time spent on each kernel
    if (PROFILING == 1 && verbose == 1) {
        double t_sum = t_calcParticleCellID + t_thrustsort + t_reorderArrays +
            t_topology + t_interact + t_bondsLinear + t_latticeBoltzmannD3Q19 +
            t_integrate + t_summation + t_integrateWalls + t_findPorositiesDev +
            t_findNSstressTensor +
            t_findNSdivphiviv + t_findNSdivtau + t_findPredNSvelocities +
            t_setNSepsilon + t_setNSdirichlet + t_setNSghostNodesDev +
            t_findNSforcing + t_jacobiIterationNS + t_updateNSvelocityPressure;

        cout << "\nKernel profiling statistics:\n"
            << "  - calcParticleCellID:\t\t" << t_calcParticleCellID/1000.0
            << " s"
            << "\t(" << 100.0*t_calcParticleCellID/t_sum << " %)\n"
            << "  - thrustsort:\t\t\t" << t_thrustsort/1000.0 << " s"
            << "\t(" << 100.0*t_thrustsort/t_sum << " %)\n"
            << "  - reorderArrays:\t\t" << t_reorderArrays/1000.0 << " s"
            << "\t(" << 100.0*t_reorderArrays/t_sum << " %)\n";
        if (params.contactmodel == 2 || params.contactmodel == 3) {
            cout
            << "  - topology:\t\t\t" << t_topology/1000.0 << " s"
            << "\t(" << 100.0*t_topology/t_sum << " %)\n";
        }
        cout << "  - interact:\t\t\t" << t_interact/1000.0 << " s"
            << "\t(" << 100.0*t_interact/t_sum << " %)\n";
        if (params.nb0 > 0) {
            cout << "  - bondsLinear:\t\t" << t_bondsLinear/1000.0 << " s"
            << "\t(" << 100.0*t_bondsLinear/t_sum << " %)\n";
        }
        cout << "  - integrate:\t\t\t" << t_integrate/1000.0 << " s"
            << "\t(" << 100.0*t_integrate/t_sum << " %)\n"
            << "  - summation:\t\t\t" << t_summation/1000.0 << " s"
            << "\t(" << 100.0*t_summation/t_sum << " %)\n"
            << "  - integrateWalls:\t\t" << t_integrateWalls/1000.0 << " s"
            << "\t(" << 100.0*t_integrateWalls/t_sum << " %)\n";
        if (navierstokes == 1) {
            cout << "  - findPorositiesDev:\t\t" << t_findPorositiesDev/1000.0
            << " s" << "\t(" << 100.0*t_findPorositiesDev/t_sum << " %)\n"
            << "  - findNSstressTensor:\t\t" << t_findNSstressTensor/1000.0
            << " s" << "\t(" << 100.0*t_findNSstressTensor/t_sum << " %)\n"
            << "  - findNSdivphiviv:\t\t" << t_findNSdivphiviv/1000.0
            << " s" << "\t(" << 100.0*t_findNSdivphiviv/t_sum << " %)\n"
            << "  - findNSdivtau:\t\t" << t_findNSdivtau/1000.0
            << " s" << "\t(" << 100.0*t_findNSdivtau/t_sum << " %)\n"
            << "  - findPredNSvelocities:\t" << t_findPredNSvelocities/1000.0
            << " s" << "\t(" << 100.0*t_findPredNSvelocities/t_sum << " %)\n"
            << "  - setNSepsilon:\t\t" << t_setNSepsilon/1000.0
            << " s" << "\t(" << 100.0*t_setNSepsilon/t_sum << " %)\n"
            << "  - setNSdirichlet:\t\t" << t_setNSdirichlet/1000.0
            << " s" << "\t(" << 100.0*t_setNSdirichlet/t_sum << " %)\n"
            << "  - setNSghostNodesDev:\t\t" << t_setNSghostNodesDev/1000.0
            << " s" << "\t(" << 100.0*t_setNSghostNodesDev/t_sum << " %)\n"
            << "  - findNSforcing:\t\t" << t_findNSforcing/1000.0 << " s"
            << "\t(" << 100.0*t_findNSforcing/t_sum << " %)\n"
            << "  - jacobiIterationNS:\t\t" << t_jacobiIterationNS/1000.0 << " s"
            << "\t(" << 100.0*t_jacobiIterationNS/t_sum << " %)\n"
            << "  - updateNSvelocityPressure:\t"
            << t_updateNSvelocityPressure/1000.0 << " s"
            << "\t(" << 100.0*t_updateNSvelocityPressure/t_sum << " %)\n";
        }
    }

    // Free GPU device memory  
    freeGlobalDeviceMemory();
    checkForCudaErrorsIter("After freeGlobalDeviceMemory()", iter);

    // Free contact info arrays
    delete[] k.contacts;
    delete[] k.distmod;
    delete[] k.delta_t;

    if (navierstokes == 1) {
        endNS();
    }

    hipDeviceReset();
}
// vim: tabstop=8 expandtab shiftwidth=4 softtabstop=4
